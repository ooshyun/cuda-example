

#include <hip/hip_runtime.h>
#include <cstdint>
#include <stdexcept>
#include <iostream>

//------------------------------------------------------------------------------
// The kernel
//------------------------------------------------------------------------------
__global__
void transform(uint32_t *output, const uint32_t *input, uint32_t size) {
  uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  if(idx >= size)
    return;

  output[idx] = input[idx] + 1;
}

//------------------------------------------------------------------------------
// Start the show
//------------------------------------------------------------------------------
int main(int argc, char **argv) {
  const uint32_t size = 5;
  const uint32_t memsize = size*4;
  uint32_t input[] = { 1, 2, 3, 4, 5 };
  uint32_t output[size];

  uint32_t *d_input;
  uint32_t *d_output;

  //----------------------------------------------------------------------------
  // Allocate memory
  //----------------------------------------------------------------------------
  auto status = hipMalloc(&d_input, memsize);
  if(status != hipSuccess)
    throw std::runtime_error("Unable to allocate GPU memory for input data");

  status = hipMalloc(&d_output, memsize);
  if(status != hipSuccess) {
   hipFree(d_input);
   throw std::runtime_error("Unable to allocate GPU memory for output data");
  }

  //----------------------------------------------------------------------------
  // Do memory copies and run the kernel
  //----------------------------------------------------------------------------
  hipMemcpy(d_input, (void*)input, memsize, hipMemcpyHostToDevice);

  uint32_t blocks = size/1024 + 1;
  transform<<<blocks, 1024>>>(d_output, d_input, size);

  hipMemcpy((void*)output, d_output, memsize, hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);

  //----------------------------------------------------------------------------
  // Print the input and the output
  //----------------------------------------------------------------------------
  std::cout << "Input: ";
  for(auto a: input)
    std::cout << a << ", ";
  std::cout << std::endl;

  std::cout << "Output: ";
  for(auto a: output)
    std::cout << a << ", ";
  std::cout << std::endl;

  return 0;
}
